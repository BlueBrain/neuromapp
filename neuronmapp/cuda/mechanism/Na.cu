#include "hip/hip_runtime.h"
#include "Na_kernels.h"
#include <sys/time.h>

__global__ void Na_state_kernel(double* __restrict p_0, double* __restrict p_1, double* __restrict p_2, double* __restrict p_3,
                double* __restrict pion_data0, int * __restrict ppvar0, double* __restrict pVEC_V, size_t size)
{
        unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < size) {
            NA_STATE(p_0, p_1, p_2, p_3, pion_data0, ppvar0, pVEC_V, idx);
        }

        __syncthreads();
}



void Na_state_wrapper(double* __restrict p_0, double* __restrict p_1, double* __restrict p_2, double* __restrict p_3,
                double* __restrict pion_data0, int * __restrict ppvar0, double* __restrict pVEC_V, size_t size)
{
        dim3 block(512);
        dim3 grid(size/block.x + 1);

        struct timeval start, stop;
        gettimeofday(&start, 0);

        Na_state_kernel<<<grid, block>>>(p_0, p_1, p_2, p_3, pion_data0, ppvar0, pVEC_V, size);

        hipDeviceSynchronize();

        gettimeofday(&stop, 0);

        printf("GPU State %f s\n", 1e-6 * (stop.tv_usec - start.tv_usec) + (float) (stop.tv_sec - start.tv_sec));
}


__global__ void Na_current_kernel(double* __restrict p_0, double* __restrict p_1, double* __restrict p_2, double* __restrict p_3,
                int * __restrict ppvar0, int * __restrict ppvar1, int * __restrict ppvar2, double* __restrict pion_data0,
                double* __restrict pion_data1, double* __restrict pion_data2, double* __restrict pVEC_V,
                double* __restrict pVEC_RHS, int * __restrict pni, size_t size)
{
        unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < size) {
            NA_CURRENT(p_0, p_1, p_2, p_3, ppvar0, ppvar1, ppvar2, pion_data0, pion_data1, pion_data2, pVEC_V, pVEC_RHS, pni, idx);
        }

        //__syncthreads();

        // Implementation for 1 GPU thread
        //for (int i = 0; i < size; i++) {
        //    NA_CURRENT(p_0, p_1, p_2, p_3, ppvar0, ppvar1, ppvar2, pion_data0, pion_data1, pion_data2, pVEC_V, pVEC_RHS, pni, i);
        //}

        //__syncthreads();

        // Implementation for 1 thread block
        //int stride = size / blockDim.x;
        //int first = idx * stride;
        //int last = (idx+1) * stride;
        //for (int i = first; i < last; i++) {
        //    NA_CURRENT(p_0, p_1, p_2, p_3, ppvar0, ppvar1, ppvar2, pion_data0, pion_data1, pion_data2, pVEC_V, pVEC_RHS, pni, i);
        //}

        //__syncthreads();

        //if (stride * blockDim.x < size) {
        //    idx = blockDim.x * stride + idx;

        //    if (idx < size) {
        //        NA_CURRENT(p_0, p_1, p_2, p_3, ppvar0, ppvar1, ppvar2, pion_data0, pion_data1, pion_data2, pVEC_V, pVEC_RHS, pni, idx);
        //    }

        //}

        //__syncthreads();
}

void Na_current_wrapper(double* __restrict p_0, double* __restrict p_1, double* __restrict p_2, double* __restrict p_3,
                int * __restrict ppvar0, int * __restrict ppvar1, int * __restrict ppvar2, double* __restrict pion_data0,
                double* __restrict pion_data1, double* __restrict pion_data2, double* __restrict pVEC_V,
                double* __restrict pVEC_RHS, int * __restrict pni, size_t size)
{
        dim3 block(512);
        dim3 grid(size/block.x + 1);
        //dim3 block(1);
        //dim3 grid(1);

        struct timeval start, stop;
        gettimeofday(&start, 0);

        Na_current_kernel<<<grid, block>>>(p_0, p_1, p_2, p_3, ppvar0, ppvar1, ppvar2, pion_data0, pion_data1, pion_data2, pVEC_V, pVEC_RHS, pni, size);

        hipDeviceSynchronize();

        gettimeofday(&stop, 0);

        printf("GPU Current %f s\n", 1e-6 * (stop.tv_usec - start.tv_usec) + (float) (stop.tv_sec - start.tv_sec));
}
